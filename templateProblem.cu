/*
 * templateProblem.cu
 *
 *  Created on: Jul 31, 2015
 *      Author: nvparrish
 */
#include "MyClass.h"
#include "kernel.cu.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

int main(int argc, char* argv[]){
	int dataSize = 1024;

	// Create device values for the input and output values
	float* d_floatData;
	float* d_floatDataResult;
	hipMalloc((void**)&d_floatData, sizeof(float) * dataSize );
	hipMalloc((void**)&d_floatDataResult, sizeof(float) * dataSize );

	// Generate random values
	hiprandState *d_state;
	hipMalloc((void**)&d_state, dataSize);
	init_rand<<<dataSize/256, 256>>>(d_state);
	make_rand<<<dataSize/256, 256>>>(d_state, d_floatData);

	// Instantiate the class
	MyClass powerClass(5);
	powerClass.calcPower<float>(d_floatData, dataSize, d_floatDataResult);

	// Free all the memory allocated with hipMalloc
	hipFree(d_state);
	hipFree(d_floatData);
	hipFree(d_floatDataResult);
}



