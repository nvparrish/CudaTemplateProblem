/*
 * templateProblem.cu
 *
 *  Created on: Jul 31, 2015
 *      Author: nvparrish
 */
#include "MyClass.h"
#include "kernel.cu.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

int main(int argc, char* argv[]){
	int dataSize = 1024;

	// Create device values for the input and output values
	float* d_floatData;
	float* d_floatDataResult;
	hipMalloc((void**)&d_floatData, sizeof(float) * dataSize );
	hipMalloc((void**)&d_floatDataResult, sizeof(float) * dataSize );

	// Generate random values
	hiprandState *d_state;
	hipMalloc((void**)&d_state, dataSize);
	init_rand<<<dataSize/256, 256>>>(d_state);
	make_rand<<<dataSize/256, 256>>>(d_state, d_floatData);

	// Instantiate the class
	MyClass powerClass(5);
	powerClass.calcPower<float>(d_floatData, dataSize, d_floatDataResult);

	// Copy the data back to take a look at it
	float* data;
	data = (float*)malloc(sizeof(float)*dataSize);
	hipMemcpy(data, d_floatDataResult, sizeof(float)*dataSize, hipMemcpyDeviceToHost);
	for(int ii = 0; ii < 128; ii++){
		for(int jj = 0; jj < 8; jj++){
			std::cout << data[ii<<3+jj] << " ";
		}
		std::cout << std::endl;
	}

	// Free all the memory allocated with hipMalloc
	hipFree(d_state);
	hipFree(d_floatData);
	hipFree(d_floatDataResult);
}



