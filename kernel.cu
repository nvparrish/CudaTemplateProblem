#include "hip/hip_runtime.h"
/*
 * kernel.cu
 *
 *  Created on: Jul 31, 2015
 *      Author: nvparrish
 */

#include "kernel.cu.h"

template<typename T>
__global__ void gpuCalcPower(T *d_data, int size, int power, T *d_out){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= size) return;

	if(power < 1){
		d_out[idx] = -1;
		return;
	}

	T value = d_data[idx];
	T product = value;
	for(int ii = 1; ii < power; ii++){
		product *= value;
	}

	d_out[idx] = product;

	return;
}

template<typename T>
void gpuCalcPowerWrapper(int blockNum, int blockSize,
T *d_data, int size, int power, T* d_out){
	gpuCalcPower<<<blockNum, blockSize>>>(d_data, size, power, d_out);
	return;
}

__global__ void init_rand(hiprandState *state){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(1337, idx, 0, &state[idx]);
}

__global__ void make_rand(hiprandState *state, float *randArray){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	randArray[idx] = hiprand_uniform(&state[idx]);
}
